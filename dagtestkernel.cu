#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <stdio.h>
#include "common.h"
#include "dagtestkernel.h"
#include "keccak.cuh"


__global__ void kernel()
{
	uint64_t nonce = 0;
	uint2 state[12];
	//vectorize nonce
    asm("mov.b64 {%0,%1},%2; \n\t" : "=r"(state[4].x), "=r"(state[4].y) : "l"(nonce));
    //execute keccak
	keccak_f1600_init(state);
	keccak_f1600_final(state);
	//dag access
	//d_dag[0].uint4s[0];
}

__global__ void genDag()
{
	//genDag
	for(int i = 0; i < d_dag_size; i++)
		for(int q = 0; q < 4; q++)
			d_dag[i].uint4s[q] = vectorize4(i*q, q*q);
	//genHeader
	for(int i = 0; i < 32 / sizeof(uint4); i++)
    	d_header.uint4s[i] = vectorize4(0, i);
}

void set_constants(
	hash128_t* _dag,
	uint32_t _dag_size,
	hash32_t _header
	)
{
	hipMemcpyToSymbol(HIP_SYMBOL(d_dag), &_dag, sizeof(hash128_t*));
	hipMemcpyToSymbol(HIP_SYMBOL(d_dag_size), &_dag_size, sizeof(uint32_t));
	hipMemcpyToSymbol(HIP_SYMBOL(d_header), &_header, sizeof(hash32_t));
}


void perf_test_gpu()
{	
	uint32_t dagSize = 128*1024*1024;
    hash128_t* dag;
    hipMalloc(reinterpret_cast<void**>(&dag), dagSize);
    hash32_t header;
    hipMalloc(reinterpret_cast<void**>(&header), dagSize);
    set_constants(dag,dagSize,header);
	genDag <<<1,1>>> ();
	hipDeviceSynchronize();
	kernel <<<1,1>>> ();
	hipDeviceSynchronize();
}
